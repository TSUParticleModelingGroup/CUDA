#include "hip/hip_runtime.h"
//General CPU code. Run on the upper trianglar part of the force matrix.
//Initail conditions are setup in a cube.																																												
// nvcc nBodyCPU.cu -o nBodyCPU -lglut -lm -lGLU -lGL
//To stop hit "control c" in the window you launched it from.

#include <sys/time.h>
#include <GL/glut.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#define N 8000

#define XWindowSize 1000
#define YWindowSize 1000

#define DRAW 1000
#define DAMP 0.5

#define DT 0.001
#define STOP_TIME 2.0

#define G 1.0
#define H 1.0

#define EYE 10.0
#define FAR 50.0

// Globals
float4 Position[N], Velocity[N], Force[N]; 

void set_initail_conditions()
{
	int i,j,k,num,particles_per_side;
    	float position_start, temp;
    	float initail_seperation;
	
	temp = pow((float)N,1.0/3.0) + 0.99999;
	particles_per_side = temp;
    	position_start = -(particles_per_side -1.0)/2.0;
	initail_seperation = 2.0;
	
	num = 0;
	for(i=0; i<particles_per_side; i++)
	{
		for(j=0; j<particles_per_side; j++)
		{
			for(k=0; k<particles_per_side; k++)
			{
			    if(N <= num) break;
				Position[num].x = position_start + i*initail_seperation;
				Position[num].y = position_start + j*initail_seperation;
				Position[num].z = position_start + k*initail_seperation;
				Position[num].w = 1.0; //mass
				
				Velocity[num].x = 0.0;
				Velocity[num].y = 0.0;
				Velocity[num].z = 0.0;
				num++;
			}
		}
	}
}

void draw_picture()
{
	int i;
	
	glClear(GL_COLOR_BUFFER_BIT);
	glClear(GL_DEPTH_BUFFER_BIT);
	
	glColor3d(1.0,1.0,0.5);
	for(i=0; i<N; i++)
	{
		glPushMatrix();
		glTranslatef(Position[i].x, Position[i].y, Position[i].z);
		glutSolidSphere(0.1,20,20);
		glPopMatrix();
	}
	
	glutSwapBuffers();
}

float4 getBodyBodyForce(float4 posMe, float4 posYou)
{
	float4 forceYouOnMe;
	float dx = posYou.x - posMe.x;
	float dy = posYou.y - posMe.y;
	float dz = posYou.z - posMe.z;
	float r2 = dx*dx + dy*dy + dz*dz;
	float r = sqrt(r2);

	float forceMag  = (G*posMe.w*posYou.w)/(r2) - (H*posMe.w*posYou.w)/(r2*r2);

	forceYouOnMe.x = forceMag*dx/r;
	forceYouOnMe.y = forceMag*dy/r;
	forceYouOnMe.z = forceMag*dz/r;

	return(forceYouOnMe);
}

void getForces()
{
	float4 forceYouOnMe; 
	
	for(int i=0; i<N; i++)
	{
		Force[i].x = 0.0;
		Force[i].y = 0.0;
		Force[i].z = 0.0;
	}

	for(int i=0; i<N; i++)
	{
		for(int j=i+1; j<N; j++)
		{
			forceYouOnMe = getBodyBodyForce(Position[i], Position[j]);
			Force[i].x += forceYouOnMe.x;
			Force[i].y += forceYouOnMe.y;
			Force[i].z += forceYouOnMe.z;
			Force[j].x -= forceYouOnMe.x;
			Force[j].y -= forceYouOnMe.y;
			Force[j].z -= forceYouOnMe.z;
		}
	}
}

void moveBodies(float time)
{
	for(int i=0; i<N; i++)
	{
		if(time == 0.0)
		{
			Velocity[i].x += ((Force[i].x-DAMP*Velocity[i].x)/Position[i].w)*0.5*DT;
			Velocity[i].y += ((Force[i].y-DAMP*Velocity[i].y)/Position[i].w)*0.5*DT;
			Velocity[i].z += ((Force[i].z-DAMP*Velocity[i].z)/Position[i].w)*0.5*DT;
		}
		else
		{
			Velocity[i].x += ((Force[i].x-DAMP*Velocity[i].x)/Position[i].w)*DT;
			Velocity[i].y += ((Force[i].y-DAMP*Velocity[i].y)/Position[i].w)*DT;
			Velocity[i].z += ((Force[i].z-DAMP*Velocity[i].z)/Position[i].w)*DT;
		}

		Position[i].x += Velocity[i].x*DT;
		Position[i].y += Velocity[i].y*DT;
		Position[i].z += Velocity[i].z*DT;
	}
}

void n_body()
{	
	int    tdraw = 0; 
	float  time = 0.0;
	
	while(time < STOP_TIME)
	{
		getForces();
		moveBodies(time);

		if(tdraw == DRAW) 
		{
			draw_picture();
			printf("\n Time = %f \n", time);
			tdraw = 0;
		}
		time += DT;
		tdraw++;
	}
}

void control()
{	
	timeval start, end;
	double totalRunTime;

	set_initail_conditions();
	draw_picture();
	
	gettimeofday(&start, NULL);
    	n_body();
    	gettimeofday(&end, NULL);
    	
    	totalRunTime = (end.tv_sec * 1000000.0 + end.tv_usec) - (start.tv_sec * 1000000.0 + start.tv_usec);
	printf("\n Totl run time = %5.15f seconds\n", (totalRunTime/1000000.0));
	
	printf("\n DONE \n");
	exit(0);
}

void Display(void)
{
	gluLookAt(EYE, EYE, EYE, 0.0, 0.0, 0.0, 0.0, 1.0, 0.0);
	glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
	control();
}

void reshape(int w, int h)
{
	glViewport(0, 0, (GLsizei) w, (GLsizei) h);

	glMatrixMode(GL_PROJECTION);

	glLoadIdentity();

	glFrustum(-0.2, 0.2, -0.2, 0.2, 0.2, FAR);

	glMatrixMode(GL_MODELVIEW);
}

int main(int argc, char** argv)
{
	glutInit(&argc,argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_DEPTH | GLUT_RGB);
	glutInitWindowSize(XWindowSize,YWindowSize);
	glutInitWindowPosition(0,0);
	glutCreateWindow("n Body CPU");
	GLfloat light_position[] = {1.0, 1.0, 1.0, 0.0};
	GLfloat light_ambient[]  = {0.0, 0.0, 0.0, 1.0};
	GLfloat light_diffuse[]  = {1.0, 1.0, 1.0, 1.0};
	GLfloat light_specular[] = {1.0, 1.0, 1.0, 1.0};
	GLfloat lmodel_ambient[] = {0.2, 0.2, 0.2, 1.0};
	GLfloat mat_specular[]   = {1.0, 1.0, 1.0, 1.0};
	GLfloat mat_shininess[]  = {10.0};
	glClearColor(0.0, 0.0, 0.0, 0.0);
	glShadeModel(GL_SMOOTH);
	glColorMaterial(GL_FRONT, GL_AMBIENT_AND_DIFFUSE);
	glLightfv(GL_LIGHT0, GL_POSITION, light_position);
	glLightfv(GL_LIGHT0, GL_AMBIENT, light_ambient);
	glLightfv(GL_LIGHT0, GL_DIFFUSE, light_diffuse);
	glLightfv(GL_LIGHT0, GL_SPECULAR, light_specular);
	glLightModelfv(GL_LIGHT_MODEL_AMBIENT, lmodel_ambient);
	glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
	glMaterialfv(GL_FRONT, GL_SHININESS, mat_shininess);
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);
	glEnable(GL_COLOR_MATERIAL);
	glEnable(GL_DEPTH_TEST);
	glutDisplayFunc(Display);
	glutReshapeFunc(reshape);
	glutMainLoop();
	return 0;
}
